#include <iostream>
#include <string>
#include <cstring>
#include <ctime>   
#include <chrono>
#include <fstream>
#include "AES.cuh"
#include <hip/hip_runtime.h>


int main(){
    unsigned char* message = readFile();
    unsigned char key[16] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16};
    unsigned char* d_key;


    int originalLen = strlen((const char*)message);
    int lenOfPaddingMessage = originalLen;
    if(lenOfPaddingMessage % 16 != 0)
        lenOfPaddingMessage = (lenOfPaddingMessage / 16 + 1) * 16;

    unsigned char* paddedMessage = new unsigned char[lenOfPaddingMessage];
    unsigned char* d_paddedMessage;
    for(int i = 0; i < lenOfPaddingMessage; i++){
        if(i >= originalLen)
            paddedMessage[i] = 0;
        else
            paddedMessage[i] = message[i];
    }

    hipMalloc((void**)&d_key, 16);
    hipMalloc((void**)&d_paddedMessage, lenOfPaddingMessage);


    hipMemcpy(d_key, key, 16, hipMemcpyHostToDevice);
    hipMemcpy(d_paddedMessage, paddedMessage, lenOfPaddingMessage, hipMemcpyHostToDevice);

    AES_CUDA(d_paddedMessage, d_key, lenOfPaddingMessage);

    hipMemcpy(paddedMessage, d_paddedMessage, lenOfPaddingMessage, hipMemcpyDeviceToHost);
    for(int i=0; i < lenOfPaddingMessage; i++){
        printHex(paddedMessage[i]);
        std::cout << " ";
    }

    delete [] paddedMessage;

    
    return 0;
}