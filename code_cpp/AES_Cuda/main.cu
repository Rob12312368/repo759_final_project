#include <iostream>
#include <string>
#include <cstring>
#include <ctime>   
#include <chrono>
#include <fstream>
#include "AES.cuh"
#include <hip/hip_runtime.h>


int main(int argc, char* argv[]){
    int thread_num = std::stoi(argv[1]);

    unsigned char* message = readFile();
    unsigned char key[16] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16};
    unsigned char* d_key;


    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    int originalLen = strlen((const char*)message);
    int lenOfPaddingMessage = originalLen;
    if(lenOfPaddingMessage % 16 != 0)
        lenOfPaddingMessage = (lenOfPaddingMessage / 16 + 1) * 16;

    unsigned char* paddedMessage = new unsigned char[lenOfPaddingMessage];
    unsigned char* d_paddedMessage;
    for(int i = 0; i < lenOfPaddingMessage; i++){
        if(i >= originalLen)
            paddedMessage[i] = 0;
        else
            paddedMessage[i] = message[i];
    }

    hipMalloc((void**)&d_key, 16);
    hipMalloc((void**)&d_paddedMessage, lenOfPaddingMessage);


    hipMemcpy(d_key, key, 16, hipMemcpyHostToDevice);
    hipMemcpy(d_paddedMessage, paddedMessage, lenOfPaddingMessage, hipMemcpyHostToDevice);

    hipEventRecord(startEvent, 0);
    AES_CUDA(d_paddedMessage, d_key, lenOfPaddingMessage, thread_num);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);

    hipMemcpy(paddedMessage, d_paddedMessage, lenOfPaddingMessage, hipMemcpyDeviceToHost);
    std::cout << "time: " << elapsedTime << "\n";
    for(int i=0; i < lenOfPaddingMessage; i++){
        printHex(paddedMessage[i]);
        std::cout << " ";
    }
    std::cout << "\n";
    delete [] paddedMessage;

    
    return 0;
}